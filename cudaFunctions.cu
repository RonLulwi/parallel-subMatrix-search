#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cudaFunctions.h"

void checkCuda(hipError_t err);

void sendImageToGPU(int** d_image, Image* image)
{
	hipError_t err;
	
	size_t imageSize = image->dim * image->dim * sizeof(int);
	
    	// allocate image memory on GPU
    	err = hipMalloc((void **)d_image, imageSize);
    	checkCuda(err);
    	
    	// copy image from host to device
    	err = hipMemcpy(*d_image, image->image, imageSize, hipMemcpyHostToDevice);
    	checkCuda(err);
}

void freeImageOnGPU(int* d_image)
{
	hipError_t err = hipFree(d_image);
	checkCuda(err);
}
	


__global__ void kernel(int* image, int imageDim, int* object, int objectDim, double threshold, int* results)
{
	double matching = 0;
	double diff;
	int p;
	int o;
	int k = 0;
	int edge = imageDim - objectDim + 1;
	
	// each thread will check if the object is in a specific location in the image (calculate the matching value) 
 	
 	int myPos = blockDim.x * blockIdx.x + threadIdx.x;
	int row = myPos / imageDim;
 	int col = myPos % imageDim;
	
	if (row < edge && col < edge) // check if my position can consist the object 
	{
		int* myImage = image + myPos;
		for (int i = 0; i < objectDim && matching <= threshold; i++)
		{
			for (int j = 0; j < objectDim && matching <= threshold; j++, k++)
			{
				p = *(myImage + imageDim *  i + j);
				o = (object)[k];
				diff = abs(p-o) / (double) p;
				matching = matching + diff;
			}
		}
		
	 	results[myPos] = (matching <= threshold);
 	}
}


int computeOnGPU(int* d_image, int imageDim, Image* object, double threshold, Position* pos)
{
	// Error code to check return values for CUDA calls
    	hipError_t err = hipSuccess;
    	
    	int objectDim = object->dim;
    	
    	int imagePixels = imageDim * imageDim;
    	int objectPixels = objectDim * objectDim;
    	
    	int* d_object;
    	int* d_result;
    	
    	size_t imageSize = imagePixels * sizeof(int);
    	size_t objectSize = objectPixels * sizeof(int);
    	
	// Allocate memory on GPU 
	allocateMemoryOnGPU(&d_object, objectSize, &d_result, imageSize);
   	
    	// Copy object from host to GPU memory
    	copyDataFromHostToDevice(d_object, object->image, objectSize);
    	
    	// allocate results array on host
   	int* result = (int*) malloc(imageSize * sizeof(int));
   	if (!result)
   	{
   		fprintf(stderr, "Problem to allocate memory\n");
        	exit(EXIT_FAILURE);
   	}
   	
   	
   	// lauch the kernel for searching object in image
   	int threadsPerBlock = 256;
    	int blocksPerGrid = (imagePixels / threadsPerBlock) + ((imagePixels % threadsPerBlock) != 0);
    	
    	kernel<<<blocksPerGrid, threadsPerBlock>>>(d_image, imageDim, d_object, objectDim, threshold, d_result);
    	checkCuda(hipGetLastError());

    	// Copy the  result from GPU to the host memory.
    	err = hipMemcpy(result, d_result, imageSize, hipMemcpyDeviceToHost);
    	checkCuda(err);
    	
    	// Calculate the result
    	int found = calculateResult(result, imageDim, objectDim, pos);
    		
    	// Free allocated memory 
    	if (hipFree(d_object) != hipSuccess || hipFree(d_result) != hipSuccess) 
    	{
    		fprintf(stderr,"%s\n", hipGetErrorString(err));
     	   	exit(EXIT_FAILURE);
    	}
    	
    	free(result);
    	
    	return found;	
}

void allocateMemoryOnGPU(int** d_object, size_t objectSize, int** d_result, size_t resultsSize)
{
	hipError_t err;
    	
    	// object
    	err = hipMalloc((void **)d_object, objectSize);
        checkCuda(err);
    	
    	// result array
    	err = hipMalloc((void **)d_result, resultsSize);
        checkCuda(err);
}

void copyDataFromHostToDevice(int* d_object, int* object, size_t objectSize)
{
	hipError_t err;
    	
   	// object
   	err = hipMemcpy(d_object, object, objectSize, hipMemcpyHostToDevice);
    	checkCuda(err);
}

int calculateResult(int* result, int imageDim, int objectDim, Position* pos)
{
	int edge = imageDim - objectDim + 1;
	
    	for (int i = 0; i < edge; i++)
    	{
    		for(int j =0; j < edge; j++)
    		{
    			if( *(result + (imageDim * i) + j) )
    			{
				pos->i = i;
				pos->j =j;
				return 1;
			}	
    		}
    	}
    	
    	return 0;
}

void checkCuda(hipError_t err)
{
	if (err != hipSuccess) 
        {
        	fprintf(stderr, "%s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
}


